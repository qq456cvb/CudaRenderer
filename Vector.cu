#include "hip/hip_runtime.h"
#include "Vector.cuh"
#include "Normal.cuh"

__host__ __device__ Vector::Vector() :
  x(0.0), y(0.0), z(0.0) { }
__host__ __device__ Vector::Vector(float v) :
  x(v), y(v), z(v) {}
__host__ __device__ Vector::Vector(const Vector &w) :
  x(w.x), y(w.y), z(w.z) { }
__host__ __device__ Vector::Vector(float xx, float yy, float zz) :
  x(xx), y(yy), z(zz) { }

__host__ __device__ float Vector::dot(const Vector &w) const {
  return this->x * w.x + this->y * w.y + this->z * w.z;
}

__host__ __device__ float Vector::dot(const Normal &n) const {
  return this->x * n.x + this->y * n.y + this->z * n.z;
}

__host__ __device__ Vector Vector::cross(const Normal &n) const {
  return Vector(this->y * n.z - this->z * n.y, this->z * n.x - this->x * n.z, this->x * n.y - this->y * n.x);
}


__host__ __device__ Vector Vector::cross(const Vector &w) const {
  return Vector(this->y * w.z - this->z * w.y, this->z * w.x - this->x * w.z, this->x * w.y - this->y * w.x);
}

__host__ __device__ Vector Vector::operator+(float s) const {
  return Vector(this->x + s, this->y + s, this->z + s);
}

__host__ __device__ Vector Vector::operator+(const Vector &w) const {
  return Vector(this->x + w.x, this->y + w.y, this->z + w.z);
}

__host__ __device__ Vector& Vector::operator+=(float s) {
  x += s; y += s; z += s;
  return *this;
}

__host__ __device__ Vector& Vector::operator+=(const Vector &v) {
  x += v.x; y += v.y; z += v.z;
  return *this;
}

__host__ __device__ Vector Vector::operator-(float s) const {
  return Vector(this->x - s, this->y - s, this->z - s);
}

__host__ __device__ Vector Vector::operator-(const Vector &w) const {
  return Vector(this->x - w.x, this->y - w.y, this->z - w.z);
}

__host__ __device__ Vector& Vector::operator-=(float s) {
  x -= s; y -= s; z -= s;
  return *this;
}

__host__ __device__ Vector Vector::operator*(float s) const {
  return Vector(s * this->x, s * this->y, s * this->z);
}

__host__ __device__ Vector Vector::operator*(const Vector &w) const {
  return Vector(this->x * w.x, this->y * w.y, this->z * w.z);
}

__host__ __device__ Vector& Vector::operator*=(float s) {
  x *= s; y *= s; z *= s;
  return *this;
}

__host__ __device__ Vector Vector::operator/(float s) const {
  return Vector(this->x / s, this->y / s, this->z / s);
}

__host__ __device__ Vector Vector::operator/(const Vector &w) const {
  return Vector(this->x / w.x, this->y / w.y, this->z / w.z);
}

__host__ __device__ Vector& Vector::operator/=(float s) {
  x /= s; y /= s; z /= s;
  return *this;
}

__host__ __device__ float Vector::norm() const {
  return sqrtf(this->x*this->x + this->y*this->y + this->z*this->z);
}

__host__ __device__ Vector& Vector::normalize() {
  *this /= norm();
  return *this;
}

__host__ __device__ Vector Vector::normalized() const {
  return *this / norm();
}

__host__ __device__ Vector operator-(const Vector &v) {
  return Vector(-v.x, -v.y, -v.z);
}

__host__ __device__ Vector operator*(float s, const Vector &v) {
  return v*s;
}

__host__ __device__ void coordinateSystem(const Vector &v1, Vector *v2, Vector *v3) {
  *v2 = Vector(0);
  if (v1.x == 0 && v1.x == 0)
  {
    v2->x = -v1.z;
  }
  else {
    v2->x = -v1.y;
    v2->y = v1.x;
  }
  *v3 = v1.cross(*v2);
}